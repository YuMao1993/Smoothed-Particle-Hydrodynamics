#include "hip/hip_runtime.h"
#include "SPHSimulator.cuh"
#include "SPHCommon.hpp"
#include <thrust/sort.h>
#include <thrust/device_ptr.h>

#include <fstream>
#include <iostream>
#include <string>
#include <algorithm>
//#include <omp.h>

#define SPH_TEST
#ifdef SPH_TEST
#define WIDTH		30
#define HEIGHT		30
#define DEPTH		30
#define SCALE		1.0f
#define STARTX		1.f
#define STARTY		1.f
#define STARTZ 		15.0f
#endif

#define px(id) (d_position[3*id])
#define py(id) (d_position[3*id+1])
#define pz(id) (d_position[3*id+2])
#define mass(id) (d_mass[id])
#define density(id) (d_density[id])
#define pressure(id) (d_pressure[id])

struct cuda_vec3
{
	double x;
	double y;
	double z;
};

struct cuda_fvec3
{
	float x;
	float y;
	float z;
};

// Device code
__device__
 float wPoly6(cuda_vec3 rvec, float h)
{
	float r = sqrt(rvec.x*rvec.x + rvec.y*rvec.y + rvec.z*rvec.z);
	float result = 0;
	if(r<=h)
	{
		result = (315.f/(64.f * SPHSim::PI * pow(h,9.f))) * pow(pow(h,2.f) - pow(r,2.f),3.f);
	}
	return result;
}

__device__ 
cuda_vec3 wPoly6Gradient(cuda_vec3 rvec, float h)
{
	float r = sqrt(rvec.x*rvec.x + rvec.y*rvec.y + rvec.z*rvec.z);
	cuda_vec3 result = {0, 0, 0};
	float c = -945.0f / (32.0f * SPHSim::PI * pow(h,9.f)) * pow(pow(h,2.f) - pow(r,2.f),2.f);
	if(r<=h)
	{
		result.x = c * rvec.x;
		result.y = c * rvec.y;
		result.z = c * rvec.z;
	}
	return result;
}

__device__
float wPoly6Laplacian(cuda_vec3 rvec, float h)
{
	float r = sqrt(rvec.x*rvec.x + rvec.y*rvec.y + rvec.z*rvec.z);
	float result = 0;
	if(r<=h)
	{
		result = 945.0f / (32.0f * SPHSim::PI * pow(h,9.f)) * (pow(h,2.f) - pow(r,2.f)) * (7.0f * pow(r,2.f) - 3.0f * pow(h,2.f));
	}
	return result;
}

__device__
cuda_vec3 wSpikyGradient(cuda_vec3 rvec, float h)
{
	float r = sqrt(rvec.x*rvec.x + rvec.y*rvec.y + rvec.z*rvec.z);
	cuda_vec3 result = {0, 0, 0};
	float c = ((- 45.f * pow(h-r,2.f)) / (r * SPHSim::PI * pow(h,6.f)));
	if((r<=h) && (r > SPHSim::EPSILON))
	{
		result.x = c * rvec.x;
		result.y = c * rvec.y;
		result.z = c * rvec.z;
	}
	return result;
}

__device__
float wViscosityLaplacian(cuda_vec3 rvec, float h)
{
	float r = sqrt(rvec.x*rvec.x + rvec.y*rvec.y + rvec.z*rvec.z);
	float result = 0;
	if(r<=h && r>SPHSim::EPSILON)
	{
		result = 45.f / (SPHSim::PI * pow(h,6.f)) * (h - r);
	}
	return result;
}

__device__
int particleToIndex(cuda_vec3 v, double kernel_h, int dim_x, int dim_y)
{
	return (int)floor(v.x/kernel_h) + (int)floor(v.y/kernel_h) * dim_x + (int)floor(v.z/kernel_h) * dim_x * dim_y;
}

__global__
void kernelMakeBlock(int* d_compressed_block, int* d_mask, int* d_scan, int arraySize)
{
	int idx = threadIdx.x + ((gridDim.x * blockIdx.y) + blockIdx.x) * blockDim.x;
	if (idx >= arraySize) return;
	if (d_mask[idx] == 1)
	{
		d_compressed_block[d_scan[idx]] = idx;
	}
}

__global__
void kernelSplitBlock(SPHSim::Bucket* d_bucket, int* mask, int arraySize)
{
	int idx = threadIdx.x + ((gridDim.x * blockIdx.y) + blockIdx.x) * blockDim.x;
	// printf("threadx: %d thready: %d threadz: %d\n", threadIdx.x, threadIdx.y, threadIdx.z);
	// printf("blockidx: %d, blockidxy: %d, blockIdxz: %d\n", blockIdx.x, blockIdx.y, blockIdx.z);
	if (idx >= arraySize) return;

	if (d_bucket[idx].size == 0)
	{
		mask[idx] = 0;
	}
	else
	{
		mask[idx] = 1;
	}	
}
__global__
void kernelComputeBucket(SPHSim::Bucket* d_bucket, int* d_bucket_index, int N, int arraySize)
{
	 int idx = threadIdx.x + ((gridDim.x * blockIdx.y) + blockIdx.x) * blockDim.x;
    // guard
    if ((idx >= 0) && (idx < N) && (d_bucket_index[idx] < arraySize) && (d_bucket_index[idx] >= 0))
    {
        atomicAdd(&d_bucket[d_bucket_index[idx]].size, 1);
        atomicCAS(&d_bucket[d_bucket_index[idx]].startIndex, 0, idx);
        atomicMin(&d_bucket[d_bucket_index[idx]].startIndex, idx);
    }
}

__global__
void kernelComputeIndex(double* d_position, int* d_index, int* d_bucket_index, int N,
									 float kernel_h, int dim_x, int dim_y)
{
	int id = threadIdx.x + ((gridDim.x * blockIdx.y) + blockIdx.x)*blockDim.x;
	if (id >= N) return;

	d_index[id] = id;
	cuda_vec3 v = {px(id), py(id), pz(id)};
	int index = particleToIndex(v, kernel_h, dim_x, dim_y);
	d_bucket_index[id] = index;
	// printf("particle: %d, position: %f, %f, %f index: %d kernel: %f, dim_x: %d, dim_y: %d N: %d\n", id, v.x, v.y, v.z, index, kernel_h, dim_x, dim_y, N);	
}

__global__
void kernelSortParticle(double* d_position, double* d_sorted_position, double* d_velocity, double* d_sorted_velocity, int* d_index, int N)
{
	int id = threadIdx.x + ((gridDim.x * blockIdx.y) + blockIdx.x)*blockDim.x;

	if (id >= N) return;
	d_sorted_position[3*id] = d_position[3*d_index[id]];
	d_sorted_position[3*id+1] = d_position[3*d_index[id]+1];
	d_sorted_position[3*id+2] = d_position[3*d_index[id]+2];

	d_sorted_velocity[3*id] = d_velocity[3*d_index[id]];
	d_sorted_velocity[3*id+1] = d_velocity[3*d_index[id]+1];
	d_sorted_velocity[3*id+2] = d_velocity[3*d_index[id]+2];

}

__global__
void kernelUpdateAndHandle(double* d_position, double* d_velocity, double* d_force, float* d_density,
									 int N, SPHSim::SPHConfig conf, double delta, double spanX, double spanY, double spanZ)
{
	int id = threadIdx.x + ((gridDim.x * blockIdx.y) + blockIdx.x)*blockDim.x;
	if (id >= N) return;
	
	// update position & velocity
	cuda_vec3 accel = {d_force[3*id] / d_density[id], d_force[3*id+1] / d_density[id], d_force[3*id+2] / d_density[id]};
	accel.x -= conf.damping * d_velocity[3*id];
	accel.y -= conf.damping * d_velocity[3*id+1];
	accel.z -= conf.damping * d_velocity[3*id+2];
	d_velocity[3*id] += delta * accel.x;
	d_velocity[3*id+1] += delta * accel.y;
	d_velocity[3*id+2] += delta * accel.z;
	d_position[3*id] += delta * d_velocity[3*id];
	d_position[3*id+1] += delta * d_velocity[3*id+1];
	d_position[3*id+2] += delta * d_velocity[3*id+2];

	// collision handle

	//handle boundary condition
	cuda_vec3 min = {0, 0, 0};
	cuda_vec3 max = {spanX, spanY, spanZ};
	if(d_position[3*id] < min.x) { d_position[3*id] = min.x; d_velocity[3*id] *= -(1.f-conf.damping); }
	if(d_position[3*id] > max.x) { d_position[3*id] = max.x; d_velocity[3*id] *= -(1.f-conf.damping); }
	if(d_position[3*id+1] < min.y) { d_position[3*id+1] = min.y; d_velocity[3*id+1] *= -(1.f-conf.damping); }
	if(d_position[3*id+1] > max.y) { d_position[3*id+1] = max.y; d_velocity[3*id+1] *= -(1.f-conf.damping); }
	if(d_position[3*id+2] < min.z) { d_position[3*id+2] = min.z; d_velocity[3*id+2] *= -(1.f-conf.damping); }
	if(d_position[3*id+2] > max.z) { d_position[3*id+2] = max.z; d_velocity[3*id+2] *= -(1.f-conf.damping); }

	
}

// for all particles, compute density and pressure
__global__ 
void kernelComputeDensity(double* d_position, SPHSim::Bucket* device_bucket, int* d_compressed_bucket, float* d_density, float* d_mass,
								float* d_pressure, int N, int numNonZeroBucket, SPHSim::SPHConfig conf, int dim_x, int dim_y, int arraySize)
{

	if (blockIdx.x >= numNonZeroBucket) return;
	if (threadIdx.x >= 27) return;

	int bucketId = d_compressed_bucket[blockIdx.x];
	int neighborBucketId;
	int count = threadIdx.x;

	for (int k = -1; k <= 1; k++)
	{
		for (int j = -1; j <= 1; j++)
		{
			for (int i = -1; i <= 1; i++)
			{
				if (--count < 0)
				{
					// printf("threadIdx: %d got %d, %d, %d\n", threadIdx.x, i, j, k);
					neighborBucketId = bucketId+i+j*dim_x+k*dim_x*dim_y;
					goto run;	
				}
			}
		}
	}

	run:
	for (int m = 0; m < device_bucket[bucketId].size; m++)
	{
		int particleId = device_bucket[bucketId].startIndex + m;
		// printf("particle: %d nieghborId: %d\n", particleId, neighborBucketId);
		__shared__ 
		float density;
		density = 0;
		__shared__
		double mx, my, mz, nx, ny, nz;
		mx = px(particleId);
		my = py(particleId);
		mz = pz(particleId);

		if (neighborBucketId >= arraySize || neighborBucketId < 0) continue; 
		for (int l = 0; l < device_bucket[neighborBucketId].size; l++)
		{
			int neighborId = device_bucket[neighborBucketId].startIndex + l;
			nx = px(neighborId);
			ny = py(neighborId);
			nz = pz(neighborId);
			// printf("neighbor particle %d: %f, %f, %f\n", l, px(neighborId), py(neighborId), pz(neighborId));
			cuda_vec3 v = {mx - nx, my - ny, mz - nz};
			
			float w = wPoly6(v, conf.kernel_h);
			atomicAdd(&density, mass(neighborId) * wPoly6(v, conf.kernel_h));
		}
		__syncthreads();
		float pressure = conf.k * (density - conf.density0); //max(xxx,0)??
		d_density[particleId] = density;
		d_pressure[particleId] = pressure;
		// printf("particle %d presure: %f density: %f\n", particleId, pressure, density);
	}
}
// compute force
__global__
void kernelComputeForce(double* d_position, double* d_velocity, SPHSim::Bucket* device_bucket, int* d_compressed_bucket, double* d_force, 
		float* d_density, float* d_mass, float* d_pressure, int N, int numNonZeroBucket, SPHSim::SPHConfig conf, int dim_x, int dim_y, int arraySize, 
		float gx, float gy, float gz)
{
	if (blockIdx.x >= numNonZeroBucket) return;
	if (threadIdx.x >= 27) return;

	int bucketId = d_compressed_bucket[blockIdx.x];
	int neighborBucketId;
	int count = threadIdx.x;

	for (int k = -1; k <= 1; k++)
	{
		for (int j = -1; j <= 1; j++)
		{
			for (int i = -1; i <= 1; i++)
			{
				if (--count < 0)
				{
					// printf("threadIdx: %d got %d, %d, %d\n", threadIdx.x, i, j, k);
					neighborBucketId = bucketId+i+j*dim_x+k*dim_x*dim_y;
					goto run;	
				}
			}
		}
	}

	run:
	for (int m = 0; m < device_bucket[bucketId].size; m++)
	{
		int particleId = device_bucket[bucketId].startIndex + m;

		__shared__ cuda_fvec3 fpressure, fviscosity, ftension, fgravity; 
		fpressure = {0, 0, 0};
		fviscosity = {0, 0, 0};
		ftension = {0, 0, 0};
		fgravity = {0, 0, 0};

		__shared__ float color_laplacian;
		color_laplacian = 0;
		__shared__ cuda_fvec3 color_gradient;
		color_gradient = {0, 0, 0};
		if (neighborBucketId >= arraySize || neighborBucketId < 0) continue; 
		for (int l = 0; l < device_bucket[neighborBucketId].size; l++)
		{
			int neighborId = device_bucket[neighborBucketId].startIndex + l;
			cuda_vec3 rvec = {px(particleId) - px(neighborId), py(particleId) - py(neighborId), pz(particleId) - pz(neighborId)};

			// pressure force
			cuda_vec3 wspikyGradient = wSpikyGradient(rvec, conf.kernel_h);
			float c = (mass(neighborId) / density(neighborId)) * ((pressure(particleId) + pressure(neighborId)) / 2.f);
			atomicAdd(&fpressure.x, -c * wspikyGradient.x);
			atomicAdd(&fpressure.y, -c * wspikyGradient.y);
			atomicAdd(&fpressure.z, -c * wspikyGradient.z);

			// viscosity force
			double wviscosityLaplacian = wViscosityLaplacian(rvec, conf.kernel_h);
			c = conf.miu * (mass(neighborId) / density(neighborId)) * wviscosityLaplacian;
			atomicAdd(&fviscosity.x, c * (d_velocity[3*neighborId] - d_velocity[3*particleId]));
			atomicAdd(&fviscosity.y, c * (d_velocity[3*neighborId+1] - d_velocity[3*particleId+1]));
			atomicAdd(&fviscosity.z, c * (d_velocity[3*neighborId+2] - d_velocity[3*particleId+2]));
			// printf("%f\n", c );

			// compute gradient of color field
			cuda_vec3 wpoly6gradient = wPoly6Gradient(rvec, conf.kernel_h);
			c = (mass(neighborId) / density(neighborId));
			atomicAdd(&color_gradient.x, c * wpoly6gradient.x);
			atomicAdd(&color_gradient.y, c * wpoly6gradient.y);
			atomicAdd(&color_gradient.z, c * wpoly6gradient.z);

			// compute laplacian of color filed
			double wpoly6Laplacian = wPoly6Laplacian(rvec, conf.kernel_h);
			atomicAdd(&color_laplacian, (mass(neighborId) / density(neighborId)) * wpoly6Laplacian);
		}
		__syncthreads();

		// gravity force
		float tmp = (float)conf.g * density(particleId);
		fgravity = {gx*tmp, gy*tmp, gz*tmp};

		float norm = sqrt(color_gradient.x*color_gradient.x + color_gradient.y*color_gradient.y + color_gradient.z*color_gradient.z);
		if(norm > SPHSim::EPSILON)
		{
			float inv_norm = 1.f / norm;
			float c = -conf.tensionCoe * color_laplacian * inv_norm;
			ftension.x = c * color_gradient.x; 
			ftension.y = c * color_gradient.y;
			ftension.z = c * color_gradient.z;
		}
		d_force[3*particleId] = fpressure.x + fviscosity.x + fgravity.x + ftension.x;
		d_force[3*particleId+1] = fpressure.y + fviscosity.y + fgravity.y + ftension.y;
		d_force[3*particleId+2] = fpressure.z + fviscosity.y + fgravity.z + ftension.z;

	}
}
 
// Host code
namespace SPHSim
{
	SPHSimulator::SPHSimulator(SPHConfig config)
	:
	conf(config)
	{
  		dim_x = ceil(conf.box.spanX / conf.kernel_h);
  		dim_y = ceil(conf.box.spanY / conf.kernel_h);
  	    dim_z = ceil(conf.box.spanZ / conf.kernel_h);
  		arraySize = dim_x * dim_y * dim_z;

        // printf("SpatialGrid initialized:\n");
		// printf("arraySize: %d\n", arraySize);
		// printf("grid_h: %f\n", conf.kernel_h);

		gravityDir = vec3(0.f,-1.f,0.f);
	}

	SPHSimulator::~SPHSimulator()
	{
		delete [] host_position;
		delete [] host_velocity;
		delete [] host_force;
		delete [] host_mass;

		hipFree( device_position );
		hipFree( device_velocity );
		hipFree( device_force );
		hipFree( device_mass );
		hipFree( device_pressure );
		hipFree( device_density );
		hipFree( device_index );
		hipFree( device_bucket_index );
		hipFree( device_bucket_mask );
		hipFree( device_scan );
	}

	void SPHSimulator::setup()
	{
#ifdef SPH_TEST

		double mass = 1.f;
		double start_x =  conf.box.spanX/5;
		double start_y =  conf.box.spanY/5;
		double start_z =  conf.box.spanZ/5;
		double end_x   =  (conf.box.spanX/5) * 4;
		double end_y   =  (conf.box.spanY/5) * 4;
		double end_z   =  (conf.box.spanZ/5) * 4;

		double f = 0.9;

		for (float x = start_x; x < end_x; x += conf.kernel_h * f)
			for (float y = start_y; y < end_y; y += conf.kernel_h * f)
				for (float z = start_z; z < end_z; z += conf.kernel_h * f)
					particles.push_back(SPHParticle(mass, vec3(x,y,z)));

		std::cout << "particle number:" << particles.size() << std::endl;


#endif
		N = particles.size();
		/*  <Device memory>  */
		// position
		hipMalloc((void **)&device_position, sizeof(double) * N * 3); 
		// velocity
		hipMalloc((void **)&device_velocity, sizeof(double) * N * 3);
		// force
		hipMalloc((void **)&device_force, sizeof(double) * N * 3);
		// mass
		hipMalloc((void **)&device_mass, sizeof(float) * N);
		// density
		hipMalloc((void **)&device_density, sizeof(float) * N);
		// presure
		hipMalloc((void **)&device_pressure, sizeof(float) * N);
		// index
		hipMalloc((void **)&device_index, sizeof(int) * N);
		// bucket index
		hipMalloc((void **)&device_bucket_index, sizeof(int) * N);
		// bucket
		hipMalloc((void **)&device_bucket, sizeof(Bucket) * arraySize);
		// bucket mask
		hipMalloc((void** )&device_bucket_mask, sizeof(int) * arraySize);
		// exclusive scan
		hipMalloc((void** )&device_scan, sizeof(int) * arraySize);

		/*  <Host memory>  */
		// position
		host_position = new double[N * 3];
		// velocity	
		host_velocity = new double[N * 3];
		// force
		host_force = new double[N * 3];
		// mass
		host_mass = new float[N];

		return;
	}

	void SPHSimulator::reset()
	{
		return;
	}

	void SPHSimulator::update(float delta)
	{


//TODO: for all particles, find neighborhoods
		/*
		>>>> copy host particle attributes to device
		*/
		//double t1_cpHtoD = omp_get_wtime();

		for (int i = 0; i < N; i++)
		{
			host_position[3 * i    ] = particles[i].position.x;
			host_position[3 * i + 1] = particles[i].position.y;
			host_position[3 * i + 2] = particles[i].position.z;
			host_velocity[3 * i    ] = particles[i].velocity.x;
			host_velocity[3 * i + 1] = particles[i].velocity.y;
			host_velocity[3 * i + 2] = particles[i].velocity.z;
			host_mass[i] = particles[i].mass;
		}
		hipMemcpy( device_position, host_position, sizeof(double) * N * 3, hipMemcpyHostToDevice );
		hipMemcpy( device_velocity, host_velocity, sizeof(double) * N * 3, hipMemcpyHostToDevice );
		hipMemcpy( device_mass, host_mass, sizeof(float) * N, hipMemcpyHostToDevice);

		// double t2_cpHtoD = omp_get_wtime();
		// std::cout << "Delta cp host to device : " 
		// 		<< (t2_cpHtoD - t1_cpHtoD) * 1000
		// 		<< " ms" << std::endl;

		dim3 threadsPerBlock(32, 1, 1);
		dim3 numBlocks(N/32+1, 1, 1);		
		// double t1_kernel = omp_get_wtime();

		kernelComputeIndex<<<numBlocks, threadsPerBlock>>>(device_position, device_index, device_bucket_index, N,
			 												conf.kernel_h, dim_x, dim_y);
		hipDeviceSynchronize();
		
		thrust::sort_by_key(thrust::device_pointer_cast<int>(device_bucket_index),
							thrust::device_pointer_cast<int>(device_bucket_index) + N,
							thrust::device_pointer_cast<int>(device_index));
		hipDeviceSynchronize();

		double* device_sorted_position;
		double* device_sorted_velocity;
		hipMalloc((void** )&device_sorted_position, sizeof(double) * N * 3);
		hipMalloc((void** )&device_sorted_velocity, sizeof(double) * N * 3);
		kernelSortParticle<<<numBlocks, threadsPerBlock>>>(device_position, device_sorted_position, device_velocity, device_sorted_velocity, device_index, N);
		hipDeviceSynchronize();
		hipMemcpy(device_position, device_sorted_position, sizeof(double) * N * 3, hipMemcpyDeviceToDevice);
		hipMemcpy(device_velocity, device_sorted_velocity, sizeof(double) * N * 3, hipMemcpyDeviceToDevice);
		hipFree(device_sorted_position);
		hipFree(device_sorted_velocity);

		hipMemset(device_bucket, 0, sizeof(Bucket) * arraySize);
		kernelComputeBucket<<<numBlocks, threadsPerBlock>>>(device_bucket, device_bucket_index, N, arraySize);
		hipDeviceSynchronize();

		// make bucket
		dim3 threadsPerBlock2(256, 1, 1);
		dim3 numBlocks2((arraySize/256)+1, 1, 1);		
		
		kernelSplitBlock<<<numBlocks2, threadsPerBlock2>>>(device_bucket, device_bucket_mask, arraySize);
		hipDeviceSynchronize();

		
		thrust::exclusive_scan(thrust::device_pointer_cast<int>(device_bucket_mask), 
							   thrust::device_pointer_cast<int>(device_bucket_mask) + arraySize,
							   thrust::device_pointer_cast<int>(device_scan));
		hipDeviceSynchronize();

		int numNonZeroBucket;
		hipMemcpy((int*)&numNonZeroBucket, device_scan+arraySize-1, sizeof(int), hipMemcpyDeviceToHost);
		hipMalloc((void** )&device_compressed_bucket, sizeof(int) * numNonZeroBucket);
		kernelMakeBlock<<<numBlocks2, threadsPerBlock2>>>(device_compressed_bucket, device_bucket_mask, device_scan, arraySize);
		hipDeviceSynchronize();


		int* host_compressed_bucket = new int[numNonZeroBucket];
		hipMemcpy(host_compressed_bucket, device_compressed_bucket, sizeof(int) * numNonZeroBucket, hipMemcpyDeviceToHost);

		// double t1_density = omp_get_wtime();

		dim3 threadsPerBlock3(32, 1, 1);
		dim3 numBlocks3(numNonZeroBucket, 1, 1);		
		kernelComputeDensity<<<numBlocks3, threadsPerBlock3>>>(device_position, device_bucket, device_compressed_bucket,
												device_density, device_mass, device_pressure, N, numNonZeroBucket, conf, dim_x, dim_y, arraySize);
		hipDeviceSynchronize();

		// double t2_density = omp_get_wtime();
		// std::cout << "Delta density : " 
		// 		<< (t2_density - t1_density) * 1000
		// 		<< " ms" << std::endl;

		//double t1_force = omp_get_wtime();
		kernelComputeForce<<<numBlocks3, threadsPerBlock3>>>(device_position, device_velocity, device_bucket, device_compressed_bucket,
									device_force, device_density, device_mass, device_pressure, N, numNonZeroBucket, conf, dim_x, dim_y, arraySize,
									gravityDir.x, gravityDir.y, gravityDir.z);
		hipDeviceSynchronize();
		// double t2_force = omp_get_wtime();
		// std::cout << "Delta force : " 
		// 		<< (t2_force - t1_force) * 1000
		// 		<< " ms" << std::endl;

		kernelUpdateAndHandle<<<numBlocks, threadsPerBlock>>>(device_position, device_velocity, device_force,
																					device_density, N, conf, delta, conf.box.spanX, conf.box.spanY, conf.box.spanZ);
		hipDeviceSynchronize();

		// double t2_kernel = omp_get_wtime();
		// std::cout << "Delta kernel : "
		// 		  << (t2_kernel-t1_kernel) * 1000
		// 		  << " ms" << std::endl;

		hipFree(device_compressed_bucket);
		/*
		>>>> copy devices particle attributes to host
		*/
		// double t1_cpDtoH = omp_get_wtime();
		hipMemcpy( host_position, device_position, sizeof(double) * N * 3, hipMemcpyDeviceToHost );
		hipMemcpy( host_velocity, device_velocity, sizeof(double) * N * 3, hipMemcpyDeviceToHost );

		for (int i = 0; i < particles.size(); i++)
		{
			particles[i].position.x = host_position[3 * i];
			particles[i].position.y = host_position[3 * i + 1];
			particles[i].position.z = host_position[3 * i + 2];
			particles[i].velocity.x = host_velocity[3 * i];
			particles[i].velocity.y = host_velocity[3 * i + 1];
			particles[i].velocity.z = host_velocity[3 * i + 2];

		 }
		// double t2_cpDtoH = omp_get_wtime();
		// std::cout << "Delta copy device to host : "
		// 		<< (t2_cpDtoH - t1_cpDtoH) * 1000
		// 		<< " ms" << std::endl;
	}

	void SPHSimulator::getData(float** position, int& vertexNum)
	{
		//allocate new memory
   		if(*position == NULL)
		   *position = (float*)malloc(sizeof(float) * particles.size() * 3);
		vertexNum = 0;

		for(int i=0; i<particles.size(); i++)
		{
			if(true)
			{
				(*position)[3*vertexNum+0] = particles[i].position.x;
				(*position)[3*vertexNum+1] = particles[i].position.y;
				(*position)[3*vertexNum+2] = particles[i].position.z;
				vertexNum ++;
			}
		}
		return;
	}

}
